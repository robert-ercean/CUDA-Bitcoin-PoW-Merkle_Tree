#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "utils.h"
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK_NONCE 256
#define THREADS_PER_BLOCK_MERKLE_INIT 256
#define THREADS_PER_BLOCK_MERKLE_REDUCE 128

// --- Helper Macro for CUDA Error Checking ---
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        hipDeviceReset(); // Resets the device on error
        exit(99);
    }
}

// CUDA sprintf alternative for nonce finding. Converts integer to its string representation. Returns string's length.
__device__ int intToString(uint64_t num, char* out) {
    if (num == 0) {
        out[0] = '0';
        out[1] = '\0';
        return 2;
    }

    int i = 0;
    while (num != 0) {
        int digit = num % 10;
        num /= 10;
        out[i++] = '0' + digit;
    }

    // Reverse the string
    for (int j = 0; j < i / 2; j++) {
        char temp = out[j];
        out[j] = out[i - j - 1];
        out[i - j - 1] = temp;
    }
    out[i] = '\0';
    return i;
}

// CUDA strlen implementation.
__host__ __device__ size_t d_strlen(const char *str) {
    size_t len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

// CUDA strcpy implementation.
__device__ void d_strcpy(char *dest, const char *src){
    int i = 0;
    while ((dest[i] = src[i]) != '\0') {
        i++;
    }
}

// CUDA strcat implementation.
__device__ void d_strcat(char *dest, const char *src){
    while (*dest != '\0') {
        dest++;
    }
    while (*src != '\0') {
        *dest = *src;
        dest++;
        src++;
    }
    *dest = '\0';
}

// Compute SHA256 and convert to hex
__host__ __device__ void apply_sha256(const BYTE *input, BYTE *output) {
    size_t input_length = d_strlen((const char *)input);
    SHA256_CTX ctx;
    BYTE buf[SHA256_BLOCK_SIZE];
    const char hex_chars[] = "0123456789abcdef";

    sha256_init(&ctx);
    sha256_update(&ctx, input, input_length);
    sha256_final(&ctx, buf);

    for (size_t i = 0; i < SHA256_BLOCK_SIZE; i++) {
        output[i * 2]     = hex_chars[(buf[i] >> 4) & 0x0F];  // High nibble
        output[i * 2 + 1] = hex_chars[buf[i] & 0x0F];         // Low nibble
    }
    output[SHA256_BLOCK_SIZE * 2] = '\0'; // Null-terminate
}

// Compare two hashes
__host__ __device__ int compare_hashes(BYTE* hash1, BYTE* hash2) {
    for (int i = 0; i < SHA256_HASH_SIZE; i++) {
        if (hash1[i] < hash2[i]) {
            return -1; // hash1 is lower
        } else if (hash1[i] > hash2[i]) {
            return 1; // hash2 is lower
        }
    }
    return 0; // hashes are equal
}



/* Kernel for the hashing of the initial transactions */
__global__ void initial_hash_kernel(const BYTE* transactions_device, int transaction_size_bytes, int num_transactions, BYTE* d_output_hashes) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < num_transactions) {
        const BYTE* src = transactions_device + tid * transaction_size_bytes;
        BYTE* dest = d_output_hashes + tid * SHA256_HASH_SIZE;
        
        apply_sha256(src, dest);
    }
}

/*
 * @param transaction_size_bytes Size of each transaction string in bytes including null terminator
 * @param transactions_host Pointer to the host memory containing the transaction strings
 * @param max_transactions_in_a_block Maximum number of transactions in a block
 * @param n The number of actual transactions in the block
 * @param merkle_root_host Pointer to the host memory where the Merkle root will be stored
 * */
void construct_merkle_root(int transaction_size_bytes, BYTE *transactions_host, int max_transactions_in_a_block, int transactions_count, BYTE merkle_root_host[SHA256_HASH_SIZE]) {
    /****************************** INITIAL TRANSACTION HASHING ******************************/
    BYTE *transactions_dev;
    /* Ping Pong style buffers */
    BYTE *d_hashes_ping, *d_hashes_pong;

    checkCudaErrors(hipMalloc((void**)&transactions_dev, transactions_count * transaction_size_bytes));
    checkCudaErrors(hipMemcpy(transactions_dev, transactions_host, transactions_count * transaction_size_bytes, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&d_hashes_ping, transactions_count * SHA256_HASH_SIZE));

    int threads_per_block = THREADS_PER_BLOCK_MERKLE_INIT;
    /* Ceil the number of blocks */
    int blocks_no = (transactions_count + threads_per_block - 1) / threads_per_block;

    initial_hash_kernel<<<blocks_no, threads_per_block>>>(transactions_dev, transaction_size_bytes, transactions_count, d_hashes_ping);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(transactions_dev));

    int n_hashes = transactions_count;

    /****************************** REDUCTION PHASE ******************************/

   
}

// TODO 2: Implement this function in CUDA
int find_nonce(BYTE *difficulty, uint32_t max_nonce, BYTE *block_content, size_t current_length, BYTE *block_hash, uint32_t *valid_nonce) {
    char nonce_string[NONCE_SIZE];
    /* This is the serialized CPU version, TODO the gpu version*/
    for (uint32_t nonce = 0; nonce <= max_nonce; nonce++) {
        sprintf(nonce_string, "%u", nonce);
        strcpy((char *)block_content + current_length, nonce_string);
        apply_sha256(block_content, block_hash);

        if (compare_hashes(block_hash, difficulty) <= 0) {
            *valid_nonce = nonce;
            return 0;
        }
    }

    return 1;
}

__global__ void dummy_kernel() {}

// Warm-up function
void warm_up_gpu() {
    BYTE *dummy_data;
    hipMalloc((void **)&dummy_data, 256);
    dummy_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    hipFree(dummy_data);
}
